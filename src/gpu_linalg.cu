#include "hip/hip_runtime.h"

/* File: gpu_linalg.cu */

#include "gpu_linalg.h"


/**
 * @brief Solve the linear system Ax=b using Gauss Seidel with SOR.
 * @param pointer to A matrix
 * @param pointer to x vector
 * @param pointer to b vector
 * @param flux convergence criteria
 * @param the maximum number of iterations
 */
void GPUlinearSolve(double *A, double* flux, double* source, double* flux_temp, 
                   int cx, int cy, int cz, int ng, double tol){
  
  /* Initialize variable pointers for GPU */
  double *dev_A, *dev_flux, *dev_source, *dev_flux_temp;
  
  /* Allocate and copy memory to GPU */
  HANDLE_ERROR( hipMalloc((void**)&dev_A, sizeof(double) * cx*cy*cz*ng*(ng+6)) );
  HANDLE_ERROR( hipMalloc((void**)&dev_flux, sizeof(double) * cx*cy*cz*ng) );
  HANDLE_ERROR( hipMalloc((void**)&dev_source, sizeof(double) * cx*cy*cz*ng) );
  HANDLE_ERROR( hipMalloc((void**)&dev_flux_temp, sizeof(double) * cx*cy*cz*ng) );

  HANDLE_ERROR( hipMemcpy( dev_A, A, sizeof(double) * cx*cy*cz*ng*(ng+6), hipMemcpyHostToDevice) );
  HANDLE_ERROR( hipMemcpy( dev_flux, flux, sizeof(double) * cx*cy*cz*ng, hipMemcpyHostToDevice) );
  HANDLE_ERROR( hipMemcpy( dev_source, source, sizeof(double) * cx*cy*cz*ng, hipMemcpyHostToDevice));

  int iter = 0;
  double _SOR_factor = 1.5;
  double residual;

  while (iter < 1000){

   /* Pass new flux to old flux */
    HANDLE_ERROR(hipMemcpy( flux_temp, flux, sizeof(double) * cx*cy*cz*ng, hipMemcpyDeviceToDevice));

    gaussSeidel<<<256, 256>>>(dev_A, dev_flux, dev_source, dev_flux_temp, 
                              cx, cy, cz, ng, _SOR_factor, 0);

    gaussSeidel<<<256, 256>>>(dev_A, dev_flux, dev_source, dev_flux_temp, 
                              cx, cy, cz, ng, _SOR_factor, 1);

    HANDLE_ERROR( hipMemcpy( &flux_temp, dev_flux_temp, sizeof(double) * cx*cy*cz*ng, hipMemcpyDeviceToHost) );
    residual = pow(pairwise_sum(flux_temp, cx*cy*cz*ng), 0.5) / (cx*cy*cz*ng);

    iter++;

    log_printf(NORMAL, "GS iter: %i, res: %f", iter, residual);
    
    if (residual < tol && iter > 10)
      break;
  }

  /* Retrieve flux from GPU */
  HANDLE_ERROR( hipMemcpy( &flux, dev_flux, sizeof(double) * cx*cy*cz*ng, hipMemcpyDeviceToHost) );

  /* Deallocate memory */
  HANDLE_ERROR( hipFree(dev_A) );
  HANDLE_ERROR( hipFree(dev_flux) );
  HANDLE_ERROR( hipFree(dev_source) );
  HANDLE_ERROR( hipFree(dev_flux_temp) );
  HANDLE_ERROR( hipFree(dev_residual) );

  return 0;
}
 

__global__ void gaussSeidel(double *dev_A, double* dev_flux, double* dev_source, double* dev_flux_temp,
                               int cx, int cy, int cz, int ng, double _SOR_factor, int color){

  int tid = threadIdx.x + blockIdx.x * blockDim.x;
  double val;
  int x, y, z;
  int row, cell;

  /* Iteration over red cells */
  while (tid < (cx*cy*cz*ng+(1-color))/2){

    x = (2*(tid+color) % (cx*cy)) % cx;
    y = (2*(tid+color) % (cx*cy)) / cx;
    z = (2*(tid+color) / (cx*cy));
    cell = z*cx*cy+y*cx+x;
    
    for (int g = 0; g < ng; g++){
      
      row = cell*ng + g;
      val = 0.0;
      
      /* Previous flux term */
      val += (1.0 - _SOR_factor) * dev_flux[row];
      
      /* Source term */
      val += _SOR_factor * dev_source[row] / dev_A(cell, g*(ng+6)+g+3);
      
      /* Left surface */
      if (x != 0)
        val -= _SOR_factor * dev_flux[row - ng] *
          dev_A(cell, g*(ng+6)) /
          dev_A(cell, g*(ng+6)+g+3);
      
      /* Back surface */
      if (y != 0)
        val -= _SOR_factor * dev_flux[row - cx * ng] *
          dev_A(cell, g*(ng+6)+1) /
          dev_A(cell, g*(ng+6)+g+3);
      
      /* Bottom surface */
      if (z != 0)
        val -= _SOR_factor * dev_flux[row - cx * cy * ng] *
          dev_A(cell, g*(ng+6)+2) /
          dev_A(cell, g*(ng+6)+g+3);
      
      /* Group-to-group */
      for (int e = 0; e < ng; e++){
        if (e != g)
          val -= _SOR_factor * dev_flux[cell*ng+e] *
            dev_A(cell, g*(ng+6)+3+e) /
            dev_A(cell, g*(ng+6)+g+3);
      }
      
      /* Right surface */
      if (x != cx - 1)
        val -= _SOR_factor * dev_flux[row + ng] *
          dev_A(cell, g*(ng+6)+ng+3) /
          dev_A(cell, g*(ng+6)+g+3);
      
      /* Front surface */
      if (y != cy - 1)
        val -= _SOR_factor * dev_flux[row + ng*cx] *
          dev_A(cell, g*(ng+6)+ng+4) /
          dev_A(cell, g*(ng+6)+g+3);
      
      /* Front surface */
      if (z != cz - 1)
        val -= _SOR_factor * dev_flux[row + ng*cx*cy] *
          dev_A(cell, g*(ng+6)+ng+5) /
          dev_A(cell, g*(ng+6)+g+3);
      
      dev_flux[row] = val;

      /* Store the square residual */
      if (dev_flux[row] != 0.0)
        dev_flux_temp[row] = (dev_flux[row] - dev_flux_temp[row]) / dev_flux[row] * (dev_flux[row] - dev_flux_temp[row]) / dev_flux[row];
      }
    
    tid += blockDim.x * gridDim.x;
  }
}
